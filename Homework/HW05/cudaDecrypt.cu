#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"

/* Q4 Make the search for the secret key parallel on the GPU using CUDA. */

//compute a*b mod p safely
__device__ unsigned int kmodprod(unsigned int a, unsigned int b, unsigned int p) {
    unsigned int za = a;
    unsigned int ab = 0;
    
    while (b > 0) {
        if (b%2 == 1) ab = (ab +  za) % p;
        za = (2 * za) % p;
        b /= 2;
    }
    return ab;
}

 //compute a^b mod p safely
__device__ unsigned int kmodExp(unsigned int a, unsigned int b, unsigned int p) {
    unsigned int z = a;
    unsigned int aExpb = 1;
    
    while (b > 0) {
        if (b%2 == 1) aExpb = kmodprod(aExpb, z, p);
        z = kmodprod(z, z, p);
        b /= 2;
    }
    return aExpb;
}

__global__ void kernelCompare(unsigned int *p, unsigned int *g, unsigned int *h, unsigned int *x) {
  unsigned int threadId = threadIdx.x;
  unsigned int blockId = blockIdx.x;
  unsigned int nBlock = blockDim.x;

  unsigned int id = threadId + blockId * nBlock;

  if (id < *p - 1) {
    if (kmodExp(*g, id + 1, *p) == *h) {
      *x = id + 1;
      printf("Secret key found! x = %u \n", *x);
    }
  }
}

int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */
  FILE *pkey = fopen("public_key.txt", "r");
  FILE *mes = fopen("message.txt", "r");

  fscanf(pkey, "%u%u%u%u", &n, &p, &g, &h);
  fscanf(mes, "%u", &Nints);

  unsigned int *Zmessage = (unsigned int *) malloc(Nints * sizeof(unsigned int));
  unsigned int *a = (unsigned int *) malloc(Nints * sizeof(unsigned int));

  for (unsigned int i = 0; i < Nints; i++) {
    fscanf(mes, "%u %u\n", &Zmessage[i], &a[i]);
  }

  fclose(pkey);
  fclose(mes);

  // find the secret key
  if (x==0 || modExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    double startTime = clock();
    
    unsigned int *d_p, *d_g, *d_h, *d_x;
    
    hipMalloc(&d_p, sizeof(unsigned int));
    hipMalloc(&d_g, sizeof(unsigned int));
    hipMalloc(&d_h, sizeof(unsigned int));
    hipMalloc(&d_x, sizeof(unsigned int));
    
    hipMemcpy(d_p, &p, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_g, &g, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_h, &h, sizeof(unsigned int), hipMemcpyHostToDevice);

    int Nthreads = 1024;
    int Nblocks = (p + Nthreads - 2)/Nthreads;
    kernelCompare<<<Nblocks, Nthreads>>>(d_p, d_g, d_h, d_x);

    hipDeviceSynchronize();
    hipMemcpy(&x, d_x, sizeof(unsigned int), hipMemcpyDeviceToHost);

    hipFree(d_p);
    hipFree(d_g);
    hipFree(d_h);
    hipFree(d_x);
    
    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  }

  /* Q3 After finding the secret key, decrypt the message */
  int bufferSize = 1024;
  unsigned char *message = (unsigned char *) malloc(bufferSize * sizeof(unsigned char));

  ElGamalDecrypt(Zmessage, a, Nints, p, x);
  convertZToString(Zmessage, Nints, message, Nints * (n - 1) / 8);
  printf("%s\n", message);
  
  return 0;
}
