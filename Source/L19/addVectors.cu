#include <stdio.h> 
#include <stdlib.h>
#include <time.h> 
#include <math.h>

#include "hip/hip_runtime.h"

//device function
__global__ void kernelAddVectors(int N, double *a, double *b, double *c) {

  int threadid = threadIdx.x; //thread number
  int blockid = blockIdx.x; //block number
  int Nblock = blockDim.x;  //number of threads in a block

  int id = threadid + blockid*Nblock;

  if (id < N) {
    c[id] = a[id] + b[id]; 
  } 
}


int main(int argc, char **argv) {

  // get vector size from command line argument
  int N = atoi(argv[1]);

  //seed RNG
  double seed = clock();
  srand48(seed);

  double *h_a, *h_b, *h_c; //host vectors

  // allocate storage
  h_a = (double *) malloc(N*sizeof(double));
  h_b = (double *) malloc(N*sizeof(double));
  h_c = (double *) malloc(N*sizeof(double));

  //populate a and b
  for (int n=0;n<N;n++) {
    h_a[n] = drand48();
    h_b[n] = drand48();
  }

  double hostStart = clock();

  // c = a + b
  for (int n=0;n<N;n++) {
    h_c[n] = h_a[n] + h_b[n];
  }
  
  double hostEnd = clock();
  double hostTime = (hostEnd - hostStart)/(double) CLOCKS_PER_SEC;

  size_t inputMem = 2*N*sizeof(double); //number of bytes the operation inputs
  size_t outMem   = 1*N*sizeof(double); //number of bytes the operation outputs

  size_t totalMem = (inputMem+outMem);

  printf("The host took %f seconds to add a and b \n", hostTime);
  printf("The efective bandwidth of the host was: %f GB/s\n", totalMem/(1E9*hostTime));
  
  //Device arrays
  double *d_a, *d_b, *d_c;

  //allocate memory on the Device with cudaMalloc
  hipMalloc(&d_a,N*sizeof(double));
  hipMalloc(&d_b,N*sizeof(double));
  hipMalloc(&d_c,N*sizeof(double));

  double copyStart = clock();

  //copy data from the host to the device
  hipMemcpy(d_a,h_a,N*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(d_b,h_b,N*sizeof(double),hipMemcpyHostToDevice);
  
  double copyEnd = clock();
  double copyTime = (copyEnd-copyStart)/(double)CLOCKS_PER_SEC;

  printf("It took %f seconds to copy the data to device. \n",copyTime);
  printf("The efective bandwidth of the copy was: %f GB/s\n", inputMem/(1E9*copyTime));

  //at this point the data is allocated and populated on the device

  int Nthreads = atoi(argv[2]); //get the number of threads per block from command line
  int Nblocks = (N+Nthreads-1)/Nthreads;

  double deviceStart = clock();

  kernelAddVectors <<<Nblocks ,Nthreads >>>(N, d_a, d_b, d_c);
  
  hipDeviceSynchronize();  

  double deviceEnd = clock();
  double deviceTime = (deviceEnd-deviceStart)/(double) CLOCKS_PER_SEC;

  printf("The device took %f seconds to add a and b \n", deviceTime); 
  printf("The efective bandwidth of the device was: %f GB/s\n", totalMem/(1E9*deviceTime));
  printf("The device was %f times faster\n", hostTime/deviceTime);

  copyStart = clock();
  hipMemcpy(h_c,d_c,N*sizeof(double),hipMemcpyDeviceToHost);
  copyEnd = clock();
  copyTime = (copyEnd-copyStart)/(double) CLOCKS_PER_SEC;

  printf("It took %f seconds to copy the data back to the host. \n",copyTime);
  printf("The efective bandwidth of the copy was: %f GB/s\n", outMem/(1E9*copyTime));

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  free(h_a);
  free(h_b);
  free(h_c);
}
