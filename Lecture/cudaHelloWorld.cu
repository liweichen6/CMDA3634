#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"

//This is my DEVICE function
//__global__ means this function is visible to the host
__global__ void kernelHelloWorld() {
	int thread = threadIdx.x; //Local thread number in a block
	int block = blockIdx.x; //Block number

	printf("Hello World from thread %d of block %d!\n", thread, block);
}

int main(int argc, char** argv) {
	int Nblocks = 10; //Number of blocks
	int Nthreads = 3; //Number of threads per block

	//Run the function "kernelHelloWorld" on the DEVICE
	kernelHelloWorld <<<Nblocks, Nthreads>>> ();

	//Wait for the DEVICE function to complete before moving on
	hipDeviceSynchronize();

	return 0;
}
