#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"

__global__ void reduction(int N, float *a, float* result) {

  int thread = threadIdx.x;
  int block  = blockIdx.x;
  int blockSize = blockDim.x;

  //unique global thread ID
  int id = thread + block*blockSize;

  __shared__ float s_sum[32];

  s_sum[id] = a[id]; //add the thread's id to start
    
  __syncthreads(); //make sure the write to shared is finished
  
  if (id<16) {//first half
   s_sum[id] += s_sum[id+16]; 
  }
  
  __syncthreads(); //make sure the write to shared is finished

  if (id<8) {//next half
   s_sum[id] += s_sum[id+8]; 
  }
  
  __syncthreads(); //make sure the write to shared is finished

  if (id<4) {//next half
   s_sum[id] += s_sum[id+4]; 
  }
  
  __syncthreads(); //make sure the write to shared is finished

  if (id<2) {//next half
   s_sum[id] += s_sum[id+2]; 
  }
  
  __syncthreads(); //make sure the write to shared is finished

  if (id<1) {//final piece
    s_sum[id] += s_sum[id+1];
    *result = s_sum[id];
  }
}


//perform a reduction on a vector of length N
int main (int argc, char **argv) {
  
  int N = 32;

  double seed=0;
  srand48(seed);

  //allocate memory on host
  float *h_a = (float*) malloc(N*sizeof(float));
  
  //populate with random data
  for (int n=0;n<N;n++) {
    h_a[n] = drand48();
  }
  
  //perform the reduction on host
  float h_sum = 0.;
  for (int n=0;n<N;n++) {
    h_sum += h_a[n];
  }
  
  printf("The Host's sum was %f \n", h_sum);

  float *d_a, *d_sum;
  hipMalloc(&d_a, N*sizeof(float));
  hipMalloc(&d_sum, 1*sizeof(float));

  //populate the device array with the same data as the host
  hipMemcpy(d_a,h_a,N*sizeof(float),hipMemcpyHostToDevice);
  
  //block dimensions
  dim3 B(32,1,1);

  //grid dimensions
  dim3 G((N+32-1)/32,1,1);

  reduction <<< G,B >>> (N, d_a, d_sum);

  hipMemcpy(&h_sum,d_sum,1*sizeof(float),hipMemcpyDeviceToHost);

  printf("The Device's sum was %f \n", h_sum);

  return 0;
}
